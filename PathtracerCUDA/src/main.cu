#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <cassert>
#include "Utility.h"

#include "hip/hip_runtime.h"
#include ""

#include "cuda_gl_interop.h"
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>

#include "Window.h"
#include <iostream>

#include "Ray.h"
#include "vec3.h"
#include "Hittable.h"
#include "Camera.h"
#include "Material.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__device__ vec3 random_in_hemisphere(const vec3 &normal, hiprandState &randState)
{
	vec3 in_unit_sphere = normalize(random_unit_vec(randState));
	return dot(in_unit_sphere, normal) > 0.0 ? in_unit_sphere : -in_unit_sphere;
}

__host__ __device__ float clamp(float x, float a, float b)
{
	x = x < a ? a : x;
	x = x > b ? b : x;
	return x;
}

__host__ __device__ vec3 clamp(vec3 x, vec3 a, vec3 b)
{
	return vec3(clamp(x.x, 0.0f, 1.0f), clamp(x.y, 0.0f, 1.0f), clamp(x.z, 0.0f, 1.0f));
}

__host__ __device__ vec3 saturate(vec3 x)
{
	return clamp(x, vec3(0.0f, 0.0f, 0.0f), vec3(1.0f, 1.0f, 1.0f));
}

__device__ vec3 getColor(const Ray &r, Hittable **world, hiprandState &randState)
{
	vec3 att = vec3(1.0f, 1.0f, 1.0f);
	Ray ray = r;
	for (int iteration = 0; iteration < 5; ++iteration)
	{
		HitRecord rec;
		if ((*world)->hit(ray, 0.001f, FLT_MAX, rec))
		{
			Ray scattered;
			vec3 attenuation;
			if (rec.m_material->scatter(ray, rec, randState, attenuation, scattered))
			{
				att *= attenuation;
				ray = scattered;
			}
			else
			{
				return vec3(0.0f, 0.0f, 0.0f);
			}
		}
		else
		{
			vec3 unitDir = normalize(ray.m_dir);
			float t = unitDir.y * 0.5f + 0.5f;
			vec3 c = (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
			return c * att;
		}
	}

	return vec3(0.0f, 0.0f, 0.0f);
}

__global__ void traceKernel(uchar4 *resultBuffer, float4 *accumBuffer, bool ignoreHistory, uint32_t frame, uint32_t width, uint32_t height, Hittable **world, hiprandState *randState, Camera camera)
{
	int threadIDx = threadIdx.x + blockIdx.x * blockDim.x;
	int threadIDy = threadIdx.y + blockIdx.y * blockDim.y;

	if (threadIDx >= width || threadIDy >= height)
	{
		return;
	}

	const uint32_t dstIdx = threadIDx + threadIDy * width;

	float4 inputColor4 = accumBuffer[dstIdx];
	vec3 inputColor(inputColor4.x, inputColor4.y, inputColor4.z);

	hiprandState &localRandState = randState[dstIdx];

	float u = (threadIDx + hiprand_uniform(&localRandState)) / float(width);
	float v = (threadIDy + hiprand_uniform(&localRandState)) / float(height);
	Ray r = camera.getRay(u, v, localRandState);
	vec3 color = getColor(r, world, localRandState);

	color += inputColor;

	vec3 resultColor = color / float(frame + 1.0f);
	resultColor = saturate(resultColor);
	resultColor.r = sqrt(resultColor.r);
	resultColor.g = sqrt(resultColor.g);
	resultColor.b = sqrt(resultColor.b);

	accumBuffer[dstIdx] = { color.r, color.g, color.b, 1.0f };
	resultBuffer[dstIdx] = { (unsigned char)(resultColor.x * 255.0f), (unsigned char)(resultColor.y * 255.0f) , (unsigned char)(resultColor.z * 255.0f), 255 };
}

__global__ void createWorld(Hittable **d_list, Hittable **d_world, hiprandState *randState, uint32_t listSize)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		uint32_t entityCount = 0;

		d_list[entityCount++] = new Sphere(vec3(0.0f, -1000.0f, 0.0f), 1000.0f,
			new Lambertian(vec3(0.5f, 0.5f, 0.5f)));

		for (int a = -11; a < 11; ++a)
		{
			for (int b = -11; b < 11; ++b)
			{
				auto chooseMat = hiprand_uniform(randState);
				vec3 center(a + 0.9f * hiprand_uniform(randState), 0.2f, b + 0.9f * hiprand_uniform(randState));
				if (length(center - vec3(4.0f, 0.2f, 0.0f)) > 0.9f)
				{
					if (chooseMat < 0.8f)
					{
						// diffuse
						auto albedo = random_vec(*randState) * random_vec(*randState);
						d_list[entityCount++] = new Sphere(center, 0.2f, new Lambertian(albedo));
					}
					else if (chooseMat < 0.95f)
					{
						// metal
						auto albedo = random_vec(*randState) * 0.5f + 0.5f;
						auto fuzz = hiprand_uniform(randState) * 0.5f;
						d_list[entityCount++] = new Sphere(center, 0.2f, new Metal(albedo, fuzz));
					}
					else
					{
						// glass
						d_list[entityCount++] = new Sphere(center, 0.2f, new Dielectric(1.5f));
					}
				}
			}
		}

		d_list[entityCount++] = new Sphere(vec3(0.0f, 1.0f, 0.0f), 1.0f, new Dielectric(1.5f));

		d_list[entityCount++] = new Sphere(vec3(-4.0f, 1.0f, 0.0f), 1.0f, new Lambertian(vec3(0.4f, 0.2f, 0.1f)));

		d_list[entityCount++] = new Sphere(vec3(4.0f, 1.0f, 0.0f), 1.0f, new Metal(vec3(0.7f, 0.6f, 0.5f), 0.0f));

		*d_world = new HittableList(d_list, entityCount);
	}
}

__global__ void freeWorld(Hittable **d_list, Hittable **d_world, uint32_t listSize)
{
	for (uint32_t i = 0; i < listSize; ++i)
	{
		delete d_list[i];
	}
	delete *d_world;
}

__global__ void initRandState(int width, int height, hiprandState *randState)
{
	int threadIDx = threadIdx.x + blockIdx.x * blockDim.x;
	int threadIDy = threadIdx.y + blockIdx.y * blockDim.y;

	if (threadIDx >= width || threadIDy >= height)
	{
		return;
	}

	uint32_t dstIdx = threadIDx + threadIDy * width;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984 + dstIdx, 0, 0, &randState[dstIdx]);
}

int main()
{
	Window window(1600, 900, "Pathtracer CUDA");

	uint32_t width = window.getWidth();
	uint32_t height = window.getHeight();

	GLuint pixelBufferGL = 0;
	hipGraphicsResource *pixelBufferCuda = nullptr;
	float4 *accumBuffer = nullptr;
	Hittable **d_list;
	Hittable **d_world;
	hiprandState *d_randState;
	uint32_t entityListSize = 22 * 22 + 4;

	auto radians = [](float degree)
	{
		return degree * (1.0f / 180.0f) * 3.14159265358979323846f;
	};

	vec3 lookfrom(13, 2, 3);
	vec3 lookat(0, 0, 0);
	vec3 vup(0, 1, 0);
	auto dist_to_focus = 10.0;
	auto aperture = 0.1;
	float aspectRatio = (float)width / height;

	Camera camera(lookfrom, lookat, vup, radians(20.0f), aspectRatio, aperture, dist_to_focus);

	// init opengl
	{
		if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
		{
			Utility::fatalExit("Failed to initialize GLAD!", EXIT_FAILURE);
		}

		glViewport(0, 0, width, height);
		assert(glGetError() == GL_NO_ERROR);
		glGenBuffers(1, &pixelBufferGL);
		assert(glGetError() == GL_NO_ERROR);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pixelBufferGL);
		glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * 4, nullptr, GL_STREAM_DRAW);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

		assert(glGetError() == GL_NO_ERROR);
	}

	// init cuda
	{
		checkCudaErrors(hipSetDevice(0));
		// register with cuda
		checkCudaErrors(hipGraphicsGLRegisterBuffer(&pixelBufferCuda, pixelBufferGL, cudaGraphicsMapFlagsWriteDiscard));

		// alloc memory for prng state
		checkCudaErrors(hipMalloc((void **)&d_randState, width * height * sizeof(hiprandState)));

		// init prng state
		dim3 threads(8, 8, 1);
		dim3 blocks((width + 7) / 8, (height + 7) / 8, 1);
		initRandState << <blocks, threads >> > (width, height, d_randState);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		// alloc memory for accum buffer and entities
		checkCudaErrors(hipMalloc((void **)&accumBuffer, width * height * sizeof(float4)));
		checkCudaErrors(hipMalloc((void **)&d_list, entityListSize * sizeof(Hittable *)));
		checkCudaErrors(hipMalloc((void **)&d_world, sizeof(Hittable *)));

		// create entities
		createWorld << <1, 1 >> > (d_list, d_world, d_randState, entityListSize);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		
	}

	uint32_t frame = 0;
	while (!window.shouldClose())
	{
		window.pollEvents();

		uchar4 *deviceMem;
		size_t numBytes;
		checkCudaErrors(hipGraphicsMapResources(1, &pixelBufferCuda));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&deviceMem, &numBytes, pixelBufferCuda));

		// do something with cuda
		dim3 threads(8, 8, 1);
		dim3 blocks((width + 7) / 8, (height + 7) / 8, 1);
		traceKernel << <blocks, threads >> > (deviceMem, accumBuffer, frame == 0, frame, width, height, d_world, d_randState, camera);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipGraphicsUnmapResources(1, &pixelBufferCuda));

		// render
		glClearColor(1.0f, 0.0f, 0.0f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);
		glRasterPos2i(-1, -1);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pixelBufferGL);
		glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

		assert(glGetError() == GL_NO_ERROR);

		window.present();
		window.setTitle(std::to_string(frame));
		++frame;
	}

	checkCudaErrors(hipDeviceSynchronize());

	// unregister pixel buffer
	checkCudaErrors(hipGraphicsUnregisterResource(pixelBufferCuda));

	// free cuda memory
	freeWorld << <1, 1 >> > (d_list, d_world, entityListSize);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_randState));

	// delete pixel buffer object
	glDeleteBuffers(1, &pixelBufferGL);
	assert(glGetError() == GL_NO_ERROR);

	return EXIT_SUCCESS;
}


