#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <cassert>
#include "Utility.h"

#include "hip/hip_runtime.h"
#include ""

#include "cuda_gl_interop.h"
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>

#include "Window.h"
#include <iostream>

#include "Ray.h"
#include "vec3.h"
#include "Hittable.h"
#include "Camera.h"
#include "Material.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__device__ vec3 random_in_hemisphere(const vec3 &normal, hiprandState &randState)
{
	vec3 in_unit_sphere = normalize(random_unit_vec(randState));
	return dot(in_unit_sphere, normal) > 0.0 ? in_unit_sphere : -in_unit_sphere;
}

__host__ __device__ float clamp(float x, float a, float b)
{
	x = x < a ? a : x;
	x = x > b ? b : x;
	return x;
}

__host__ __device__ vec3 clamp(vec3 x, vec3 a, vec3 b)
{
	return vec3(clamp(x.x, 0.0f, 1.0f), clamp(x.y, 0.0f, 1.0f), clamp(x.z, 0.0f, 1.0f));
}

__host__ __device__ vec3 saturate(vec3 x)
{
	return clamp(x, vec3(0.0f, 0.0f, 0.0f), vec3(1.0f, 1.0f, 1.0f));
}

__device__ vec3 getColor(const Ray &r, Hittable **world, hiprandState &randState)
{
	vec3 att = vec3(1.0f, 1.0f, 1.0f);
	Ray ray = r;
	for (int iteration = 0; iteration < 5; ++iteration)
	{
		HitRecord rec;
		if ((*world)->hit(ray, 0.001f, FLT_MAX, rec))
		{
			Ray scattered;
			vec3 attenuation;
			if (rec.m_material->scatter(ray, rec, randState, attenuation, scattered))
			{
				att *= attenuation;
				ray = scattered;
			}
			else
			{
				return vec3(0.0f, 0.0f, 0.0f);
			}
		}
		else
		{
			vec3 unitDir = normalize(ray.m_dir);
			float t = unitDir.y * 0.5f + 0.5f;
			vec3 c = (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
			return c * att;
		}
	}

	return vec3(0.0f, 0.0f, 0.0f);
}

__global__ void traceKernel(uchar4 *resultBuffer, float4 *accumBuffer, bool ignoreHistory, uint32_t frame, uint32_t width, uint32_t height, Hittable **world, hiprandState *randState)
{
	int threadIDx = threadIdx.x + blockIdx.x * blockDim.x;
	int threadIDy = threadIdx.y + blockIdx.y * blockDim.y;

	if (threadIDx >= width || threadIDy >= height)
	{
		return;
	}

	const uint32_t dstIdx = threadIDx + threadIDy * width;

	float4 inputColor4 = accumBuffer[dstIdx];
	vec3 inputColor(inputColor4.x, inputColor4.y, inputColor4.z);

	hiprandState &localRandState = randState[dstIdx];

	float u = (threadIDx + hiprand_uniform(&localRandState)) / float(width);
	float v = (threadIDy + hiprand_uniform(&localRandState)) / float(height);
	Camera cam((float)width / height);
	Ray r = cam.getRay(u, v);
	vec3 color = getColor(r, world, localRandState);

	color += inputColor;

	vec3 resultColor = color / float(frame + 1.0f);
	resultColor = saturate(resultColor);
	resultColor.r = sqrt(resultColor.r);
	resultColor.g = sqrt(resultColor.g);
	resultColor.b = sqrt(resultColor.b);

	accumBuffer[dstIdx] = { color.r, color.g, color.b, 1.0f };
	resultBuffer[dstIdx] = { (unsigned char)(resultColor.x * 255.0f), (unsigned char)(resultColor.y * 255.0f) , (unsigned char)(resultColor.z * 255.0f), 255 };
}

__global__ void createWorld(Hittable **d_list, Hittable **d_world)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		d_list[0] = new Sphere(vec3(0.0f, 0.0f, -1.0f), 0.5f,
			new Lambertian(vec3(0.7f, 0.3f, 0.3f)));
		d_list[1] = new Sphere(vec3(0.0f, -100.5f, -1.0f), 100.0f,
			new Lambertian(vec3(0.8f, 0.8f, 0.0f)));

		d_list[2] = new Sphere(vec3(1.0f, 0.0f, -1.0f), 0.5f,
			new Metal(vec3(0.8f, 0.6f, 0.2f), 1.0f));

		d_list[3] = new Sphere(vec3(-1.0f, 0.0f, -1.0f), 0.5f,
			new Metal(vec3(0.8f, 0.8f, 0.8f), 0.3f));

		*d_world = new HittableList(d_list, 4);
	}
}

__global__ void freeWorld(Hittable **d_list, Hittable **d_world)
{
	delete d_list[0];
	delete d_list[1];
	delete *d_world;
}

__global__ void initRandState(int width, int height, hiprandState *randState)
{
	int threadIDx = threadIdx.x + blockIdx.x * blockDim.x;
	int threadIDy = threadIdx.y + blockIdx.y * blockDim.y;

	if (threadIDx >= width || threadIDy >= height)
	{
		return;
	}

	uint32_t dstIdx = threadIDx + threadIDy * width;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984 + dstIdx, 0, 0, &randState[dstIdx]);
}

int main()
{
	Window window(1600, 900, "Pathtracer CUDA");

	uint32_t width = window.getWidth();
	uint32_t height = window.getHeight();

	GLuint pixelBufferGL = 0;
	hipGraphicsResource *pixelBufferCuda = nullptr;
	float4 *accumBuffer = nullptr;
	Hittable **d_list;
	Hittable **d_world;
	hiprandState *d_randState;


	// init opengl
	{
		if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
		{
			Utility::fatalExit("Failed to initialize GLAD!", EXIT_FAILURE);
		}

		glViewport(0, 0, width, height);
		assert(glGetError() == GL_NO_ERROR);
		glGenBuffers(1, &pixelBufferGL);
		assert(glGetError() == GL_NO_ERROR);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pixelBufferGL);
		glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * 4, nullptr, GL_STREAM_DRAW);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

		assert(glGetError() == GL_NO_ERROR);
	}

	// init cuda
	{
		checkCudaErrors(hipSetDevice(0));
		// register with cuda
		checkCudaErrors(hipGraphicsGLRegisterBuffer(&pixelBufferCuda, pixelBufferGL, cudaGraphicsMapFlagsWriteDiscard));
		checkCudaErrors(hipMalloc((void **)&accumBuffer, width * height * sizeof(float4)));
		checkCudaErrors(hipMalloc((void **)&d_list, 4 * sizeof(Hittable *)));
		checkCudaErrors(hipMalloc((void **)&d_world, sizeof(Hittable *)));
		createWorld << <1, 1 >> > (d_list, d_world);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipMalloc((void **)&d_randState, width * height * sizeof(hiprandState)));

		dim3 threads(8, 8, 1);
		dim3 blocks((width + 7) / 8, (height + 7) / 8, 1);
		initRandState << <blocks, threads >> > (width, height, d_randState);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
	}

	uint32_t frame = 0;
	while (!window.shouldClose())
	{
		window.pollEvents();

		uchar4 *deviceMem;
		size_t numBytes;
		checkCudaErrors(hipGraphicsMapResources(1, &pixelBufferCuda));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&deviceMem, &numBytes, pixelBufferCuda));

		// do something with cuda
		dim3 threads(8, 8, 1);
		dim3 blocks((width + 7) / 8, (height + 7) / 8, 1);
		traceKernel << <blocks, threads >> > (deviceMem, accumBuffer, frame == 0, frame, width, height, d_world, d_randState);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipGraphicsUnmapResources(1, &pixelBufferCuda));

		// render
		glClearColor(1.0f, 0.0f, 0.0f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);
		glRasterPos2i(-1, -1);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pixelBufferGL);
		glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

		assert(glGetError() == GL_NO_ERROR);

		window.present();
		window.setTitle(std::to_string(frame));
		++frame;
	}

	checkCudaErrors(hipDeviceSynchronize());

	// unregister pixel buffer
	checkCudaErrors(hipGraphicsUnregisterResource(pixelBufferCuda));

	// free cuda memory
	freeWorld << <1, 1 >> > (d_list, d_world);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_randState));

	// delete pixel buffer object
	glDeleteBuffers(1, &pixelBufferGL);
	assert(glGetError() == GL_NO_ERROR);

	return EXIT_SUCCESS;
}


